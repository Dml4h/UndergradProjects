#include <stdio.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>

__global__    // <--- writing a kernel function to be run on the gpu (called on host)
void saveIDs(int *idsOut)
{
    const int tid = threadIdx.x;  // <-- multi-dimensional; just getting one dim right now

    idsOut[tid] = tid;  // put my gpu thread id into the vector
}

int main(int argc, char *argv[])
{
    int i, numThreads;
    int *dIDs;  // d prefix implies on "device"
    int *hIDs;  // h prefix implies on "host"
	bool tFlag = false; //flag to check if command line -t present
    printf("starting\n");
	// The input will come from a file specified as the first command-line argument.
	// The second command-line arg is optional; if present, it will be  -t  and means
	// to just print the final total line (below) and NOT print all output values.
	if(argc == 2)
		printf("argc: 2 tflag %d\n", tFlag);
	else{
		tFlag = true;
		printf("argc: %d %d\n", argc, tFlag);
		
		}

    numThreads = 32;

    hipMalloc(&dIDs, sizeof(int) * numThreads);
    
	saveIDs<<< 1, numThreads >>>(dIDs);   // launch kernel; num blks and num thrds per blk
	
	hipDeviceSynchronize(); //wait for execution of kernal
    hIDs = (int *) malloc(sizeof(int) * numThreads);

    hipMemcpy(hIDs, dIDs, sizeof(int) * numThreads,  hipMemcpyDeviceToHost);

    for (i=0; i < numThreads; i++)
    {
        printf("hIds[%d]  %d\n",i,hIDs[i]);
    }

    free(hIDs);
    hipFree(dIDs);
	hipDeviceReset();
	//fclose(inFile);
    printf("done\n");

    return 0;
}

