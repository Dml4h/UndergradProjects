//Write a C program that uses  CUDA  solve the problem below:
//	This program will perform as a single layer of neurons in a neural network.
//	There will be:
//    a 1-D vector of input values
//    a 2-D array of weights
//	The program should produce the 1-D vector of output values and print them
//  UNLESS the -t cmd-line flag is specified (below).
//	The activation function is sigmoid. Doubles used instead of floats.


#include <stdio.h>
#include <stdlib.h>
// CUDA runtime
#include <hip/hip_runtime.h>

//#ifndef FOPEN
//#define FOPEN(fHandle, filename, mode) (fHandle = fopen(filename, mode))
//#endif

//void openFile(char *argv[1]){
// File *inFile = fopen(argv[1],"r");
// printf("here \n");
// fclose(inFile);
//}
__global__    // <--- writing a kernel function to be run on the gpu (called on host)
void saveIDs(int *idsOut)
{
    const int tid = threadIdx.x;  // <-- multi-dimensional; just getting one dim right now

    idsOut[tid] = tid;  // put my gpu thread id into the vector
}

int main(int argc, char *argv[])
{
    int i, numThreads;
    int *dIDs;  // d prefix implies on "device"
    int *hIDs;  // h prefix implies on "host"
	bool tFlag = false; //flag to check if command line -t present
	//File *inFile;
    printf("starting\n");
	// The input will come from a file specified as the first command-line argument.
	// The second command-line arg is optional; if present, it will be  -t  and means
	// to just print the final total line (below) and NOT print all output values.
    //File *inFile = fopen(argv[1],"r");
	//openFile(argv);
//	if(inFile == NULL){
//		perror("error opening file");
//		exit(EXIT_FAILURE);
//	}
	if(argc == 2)
		printf("argc: 2 tflag %d\n", tFlag);
	else{
		tFlag = true;
		printf("argc: %d %d\n", argc, tFlag);
		
		}
	//if (argc > 1)
    //    numThreads = atoi(argv[1]);
    //else
    numThreads = 32;

    hipMalloc(&dIDs, sizeof(int) * numThreads);
    
	//openFile<< 1, 1>>(argv);
	saveIDs<<< 1, numThreads >>>(dIDs);   // launch kernel; num blks and num thrds per blk
	
	hipDeviceSynchronize(); //wait for execution of kernal
    hIDs = (int *) malloc(sizeof(int) * numThreads);

    hipMemcpy(hIDs, dIDs, sizeof(int) * numThreads,  hipMemcpyDeviceToHost);

    for (i=0; i < numThreads; i++)
    {
        printf("hIds[%d]  %d\n",i,hIDs[i]);
    }

    free(hIDs);
    hipFree(dIDs);
	hipDeviceReset();
	//fclose(inFile);
    printf("done\n");

    return 0;
}

